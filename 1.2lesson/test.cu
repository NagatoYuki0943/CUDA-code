#include <stdio.h>
#include <hip/hip_runtime.h>



/**
* __global__：表示该函数被CPU调用启动，在GPU上执行
* __device__：表示GPU端调用且在GPU端执行的函数
* __host__：表示CPU端调用且在CPU端执行的函数，无任何修饰符时默认是CPU端函数
*/
__global__ void hello_from_gpu()
{
    printf("Hello World from the the GPU\n");
}


int main(void)
{
    hello_from_gpu<<<4, 4>>>();
    hipDeviceSynchronize();

    return 0;
}
